#include "hip/hip_runtime.h"
#include "cuda.cuh"

/* device */
__device__ void device_to_device()
{

}

/* kernel */
template<class T>
__global__ void host_to_device(T* pSrc, T* pDst)
{
	int i = GPU_THREAD_INDEX_X;
	pDst[i] = sin(pSrc[i]) * sin(pSrc[i]) + cos(pSrc[i]) * cos(pSrc[i]);
}

/* default */
__host__ void host_to_host()
{

}

template<class T>
void GPU<T>::AllocateMemory()
{
	HeavyProcess<T>::AllocateMemory();
	size_t nbytes = n * sizeof(T);
	hipMalloc(&pDevSrc, nbytes);
	hipMalloc(&pDevDst, nbytes);
	hipMemset(pDevSrc, 0, nbytes);
	hipMemset(pDevDst, 0, nbytes);
	hipMemcpy(pDevSrc, pSrc, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(pDevDst, pDst, nbytes, hipMemcpyHostToDevice);
}

template<class T>
void GPU<T>::HeavyCalc()
{
	dim3 block(128, 1, 1);
	dim3 grid(128, 1, 1);
	host_to_device<<<grid, block, 0>>>(pDevSrc, pDevDst);
}

template<class T>
void GPU<T>::CopyDstMemory()
{
	size_t nbytes = n * sizeof(T);
	hipMemcpy(pDst, pDevDst, nbytes, hipMemcpyDeviceToHost);
}

template void GPU<float>::AllocateMemory();
template void GPU<double>::AllocateMemory();
template void GPU<float>::HeavyCalc();
template void GPU<double>::HeavyCalc();
template void GPU<float>::CopyDstMemory();
template void GPU<double>::CopyDstMemory();
