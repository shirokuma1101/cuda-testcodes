#include "hip/hip_runtime.h"
#include "cuda.cuh"

#define GPU_THREAD_INDEX_X blockIdx.x * blockDim.x + threadIdx.x
#define GPU_THREAD_INDEX_Y blockIdx.y * blockDim.y + threadIdx.y
#define GPU_THREAD_INDEX_Z blockIdx.z * blockDim.z + threadIdx.z

/* device */
__device__ void device_to_device()
{

}

/* kernel */
template<class T>
__global__ void host_to_device(T* pSrc, T* pDst)
{
    int i = GPU_THREAD_INDEX_X;
    pDst[i] = sin(pSrc[i]) * sin(pSrc[i]) + cos(pSrc[i]) * cos(pSrc[i]);
}

/* default */
__host__ void host_to_host()
{

}

template<class T>
void GPU<T>::AllocateMemory()
{
    HeavyProcess<T>::AllocateMemory();

    for (auto&& e : streams) {
        auto tmp = GPUStream();
        hipStreamCreate(&tmp.cudaStream);
        size_t nbytes = e.num * sizeof(T);
        hipMalloc(&tmp.pDevSrc, nbytes);
        hipMalloc(&tmp.pDevDst, nbytes);
        hipMemset(tmp.pDevSrc, 0, nbytes);
        hipMemcpyAsync(tmp.pDevSrc, e.pSrc, nbytes, hipMemcpyHostToDevice, tmp.cudaStream);
        gpuStreams.push_back(tmp);
    }
}

template<class T>
void GPU<T>::HeavyCalc()
{
    for (size_t i = 0; i < gpuStreams.size(); ++i) {
        dim3 block(512, 1, 1);
        dim3 grid(streams[i].num / block.x, 1, 1);
        host_to_device<<<grid, block, 0, gpuStreams[i].cudaStream>>>(gpuStreams[i].pDevSrc, gpuStreams[i].pDevDst);
    }
}

template<class T>
void GPU<T>::CopyDstMemory()
{
    for (size_t i = 0; i < gpuStreams.size(); ++i) {
        size_t nbytes = streams[i].num * sizeof(T);
        hipMemcpyAsync(streams[i].pDst, gpuStreams[i].pDevDst, nbytes, hipMemcpyDeviceToHost, gpuStreams[i].cudaStream);
    }

    for (auto&& e : gpuStreams) {
        hipStreamSynchronize(e.cudaStream);
    }
}

template void GPU<float>::AllocateMemory();
template void GPU<double>::AllocateMemory();
template void GPU<float>::HeavyCalc();
template void GPU<double>::HeavyCalc();
template void GPU<float>::CopyDstMemory();
template void GPU<double>::CopyDstMemory();
